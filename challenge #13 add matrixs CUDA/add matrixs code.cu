#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

// parallel SAXPY‐style kernel (here just x + y)
__global__
void addKernel(int n, float* x, float* y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = x[i] + y[i];
    }
}

int main() {
    const int minExp = 15;
    const int maxExp = 25;
    const int threadsPerBlock = 256;

    for (int exp = minExp; exp <= maxExp; ++exp) {
        int N = 1 << exp;
        float *x, *y;

        // 1) allocate unified memory
        hipMallocManaged(&x, N * sizeof(float));
        hipMallocManaged(&y, N * sizeof(float));

        // 2) initialize on host
        for (int i = 0; i < N; ++i) {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }

        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

        // 3) setup CUDA event timers
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // 4) record, launch, record, sync
        hipEventRecord(start);
        addKernel<<<blocksPerGrid, threadsPerBlock>>>(N, x, y);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        // 5) compute elapsed time
        float ms = 0.0f;
        hipEventElapsedTime(&ms, start, stop);

        // 6) verify correctness
        float maxError = 0.0f;
        for (int i = 0; i < N; ++i) {
            maxError = fmax(maxError, fabs(y[i] - 3.0f));
        }

        // 7) print results
        std::cout
            << "2^" << exp
            << " (N=" << N << "): "
            << ms << " ms,  maxError=" << maxError
            << std::endl;

        // 8) cleanup
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(x);
        hipFree(y);
    }

    return 0;
}
