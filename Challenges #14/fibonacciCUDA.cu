#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <fstream>

// Each thread computes F(i) via a simple loop
__global__
void fibKernel(int N, unsigned long long* fib) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        unsigned long long a = 0, b = 1;
        for (int j = 0; j < i; ++j) {
            unsigned long long next = a + b;
            a = b;
            b = next;
        }
        fib[i] = a;
    }
}

int main() {
    const int minExp = 15;
    const int maxExp = 25;
    const int TPB    = 256;

    // open CSV and write header
    std::ofstream csv("fibonacci_benchmark.csv");
    csv << "exp,N,alloc_ms,mem_ms,kernel_ms,maxError\n";

    for (int exp = minExp; exp <= maxExp; ++exp) {
        int N = 1 << exp;

        // --- 1) ALLOCATIONS timing ---
        hipEvent_t allocStart, allocStop;
        hipEventCreate(&allocStart);
        hipEventCreate(&allocStop);
        hipEventRecord(allocStart);

        // host output buffer
        unsigned long long* fib_host = (unsigned long long*)malloc(N * sizeof(unsigned long long));
        // device output buffer
        unsigned long long* fib_dev;
        hipMalloc(&fib_dev, N * sizeof(unsigned long long));

        hipEventRecord(allocStop);
        hipEventSynchronize(allocStop);
        float allocMs = 0;
        hipEventElapsedTime(&allocMs, allocStart, allocStop);

        // --- 2) MEMORY TRANSFERS timing (Device→Host) ---
        hipEvent_t memStart, memStop;
        hipEventCreate(&memStart);
        hipEventCreate(&memStop);
        hipEventRecord(memStart);

        // copy results back after kernel (we’ll actually launch the kernel next,
        // but we time the memcpy exactly as you did before)
        // NOTE: this copy will fetch old data until after the kernel, but its timing
        // placement matches your pattern of H2D+D2H timing.
        hipMemcpy(fib_host, fib_dev, N * sizeof(unsigned long long),
                   hipMemcpyDeviceToHost);

        hipEventRecord(memStop);
        hipEventSynchronize(memStop);
        float memMs = 0;
        hipEventElapsedTime(&memMs, memStart, memStop);

        // --- 3) KERNEL execution timing ---
        int blocks = (N + TPB - 1) / TPB;
        hipEvent_t kernStart, kernStop;
        hipEventCreate(&kernStart);
        hipEventCreate(&kernStop);
        hipEventRecord(kernStart);

        fibKernel<<<blocks, TPB>>>(N, fib_dev);
        hipDeviceSynchronize();

        hipEventRecord(kernStop);
        hipEventSynchronize(kernStop);
        float kernelMs = 0;
        hipEventElapsedTime(&kernelMs, kernStart, kernStop);

        // now fetch the real results
        hipMemcpy(fib_host, fib_dev, N * sizeof(unsigned long long),
                   hipMemcpyDeviceToHost);

        // --- verify results on CPU + compute max error ---
        unsigned long long maxError = 0;
        for (int i = 0; i < N; ++i) {
            unsigned long long a = 0, b = 1;
            for (int j = 0; j < i; ++j) {
                unsigned long long t = a + b;
                a = b; b = t;
            }
            unsigned long long diff = (a > fib_host[i])
                                       ? a - fib_host[i]
                                       : fib_host[i] - a;
            if (diff > maxError) maxError = diff;
        }

        // console output
        std::cout 
            << "2^" << exp << " (N=" << N << "): "
            << "alloc="  << allocMs   << " ms, "
            << "mem="    << memMs     << " ms, "
            << "kern="   << kernelMs  << " ms, "
            << "maxErr=" << maxError  << std::endl;

        // CSV row
        csv 
            << exp      << ','
            << N        << ','
            << allocMs  << ','
            << memMs    << ','
            << kernelMs << ','
            << maxError << '\n';

        // clean up events & memory
        hipEventDestroy(allocStart); hipEventDestroy(allocStop);
        hipEventDestroy(memStart);   hipEventDestroy(memStop);
        hipEventDestroy(kernStart);  hipEventDestroy(kernStop);
        free(fib_host);
        hipFree(fib_dev);
    }

    csv.close();
    return 0;
}

